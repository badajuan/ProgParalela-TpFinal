

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <sys/time.h>

#define BLOCK_SIZE  16
#define HEADER_SIZE 138
#define BLOCK_SIZE_SH 18

typedef unsigned char BYTE;

/**
 * Structure that represents a BMP image.
 */
typedef struct
{
    int   ancho;
    int   alto;
    float *data;
} BMPImage;

typedef struct timeval tval;

BYTE g_info[HEADER_SIZE]; // Reference header

/**
 * Reads a BMP 24bpp file and returns a BMPImage structure.
 * Thanks to https://stackoverflow.com/a/9296467
 */
BMPImage readBMP(char *filename){
    BMPImage bitmap = { 0 };
    int      size   = 0;
    BYTE     *data  = NULL;
    FILE     *file  = fopen(filename, "rb");
    
    // Read the header (expected BGR - 24bpp)
    fread(g_info, sizeof(BYTE), HEADER_SIZE, file);

    // Get the image ancho / alto from the header
    bitmap.ancho  = *((int *)&g_info[18]);
    bitmap.alto = *((int *)&g_info[22]);
    size          = *((int *)&g_info[34]);
    
    // Read the image data
    data = (BYTE *)malloc(sizeof(BYTE) * size);
    fread(data, sizeof(BYTE), size, file);
    
    // Convert the pixel values to float
    bitmap.data = (float *)malloc(sizeof(float) * size);
    
    for (int i = 0; i < size; i++)
    {
        bitmap.data[i] = (float)data[i];
    }
    
    fclose(file);
    free(data);
    
    return bitmap;
}

/**
 * Writes a BMP file in grayscale given its image data and a filename.
 */
void writeBMPGrayscale(int ancho, int alto, float *image, char *filename){
    FILE *file = NULL;
    
    file = fopen(filename, "wb");
    
    // Write the reference header
    fwrite(g_info, sizeof(BYTE), HEADER_SIZE, file);
    
    // Unwrap the 8-bit grayscale into a 24bpp (for simplicity)
    for (int h = 0; h < alto; h++)
    {
        int offset = h * ancho;
        
        for (int w = 0; w < ancho; w++)
        {
            BYTE pixel = (BYTE)((image[offset + w] > 255.0f) ? 255.0f :
                                (image[offset + w] < 0.0f)   ? 0.0f   :
                                                               image[offset + w]);
            
            // Repeat the same pixel value for BGR
            fputc(pixel, file);
            fputc(pixel, file);
            fputc(pixel, file);
        }
    }
    
    fclose(file);
}

/**
 * Releases a given BMPImage.
 */
void freeBMP(BMPImage bitmap){
    free(bitmap.data);
}

/**
 * Checks if there has been any CUDA error. The method will automatically print
 * some information and exit the program when an error is found.
 */
void checkCUDAError(){
    hipError_t hipError_t = hipGetLastError();
    
    if(hipError_t != hipSuccess)
    {
        printf("CUDA Error: Returned %d: %s\n", hipError_t,
                                                hipGetErrorString(hipError_t));
        exit(-1);
    }
}

/**
 * Calculates the elapsed time between two time intervals (in milliseconds).
 */
double get_elapsed(tval t0, tval t1){
    return (double)(t1.tv_sec - t0.tv_sec) * 1000.0L + (double)(t1.tv_usec - t0.tv_usec) / 1000.0L;
}

/**
 * Stores the result image and prints a message.
 */
void store_result(int index, double elapsed_cpu, double elapsed_gpu,
                     int ancho, int alto, float *image){
    char path[255];
    
    sprintf(path, "images/hw3_result_%d.bmp", index);
    writeBMPGrayscale(ancho, alto, image, path);
    
    printf("Step #%d Completed - Result stored in \"%s\".\n", index, path);
    printf("Elapsed CPU: %fms / ", elapsed_cpu);
    
    if (elapsed_gpu == 0)
    {
        printf("[GPU version not available]\n");
    }
    else
    {
        printf("Elapsed GPU: %fms\n", elapsed_gpu);
    }
}

/**
 * Converts a given 24bpp image into 8bpp grayscale using the CPU.
 */
void cpu_grayscale(int ancho, int alto, float *image, float *image_out){
    for (int y = 0; y < alto; y++)
    {
        int offset_out = y * ancho;      // 1 color per pixel
        int offset     = offset_out * 3; // 3 colors per pixel
        
        for (int x = 0; x < ancho; x++)
        {
            float *pixel = &image[offset + x * 3];
            
            // Convert to grayscale following the "luminance" model
            image_out[offset_out + x] = pixel[0] * 0.0722f + // B
                                        pixel[1] * 0.7152f + // G
                                        pixel[2] * 0.2126f;  // R
        }
    }
}

/**
 * Converts a given 24bpp image into 8bpp grayscale using the GPU.
 */
__global__ void gpu_grayscale(int ancho, int alto, float *image, float *image_out){
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if( x < ancho && y < alto){
        float *pixel = &image[(y*ancho + x)*3]; //Multiplico por 3 para tener en cuenta la linealización del arreglo
        image_out[y*ancho + x]= pixel[0] * 0.0722f + // B
                                pixel[1] * 0.7152f + // G
                                pixel[2] * 0.2126f;  // R
    }    
}

/**
 * Applies a 3x3 convolution matrix to a pixel using the CPU.
 */
float cpu_applyFilter(float *image, int stride, float *matrix, int filter_dim){
    float pixel = 0.0f;
    
    for (int h = 0; h < filter_dim; h++)
    {
        int offset        = h * stride;
        int offset_kernel = h * filter_dim;
        
        for (int w = 0; w < filter_dim; w++)
        {
            pixel += image[offset + w] * matrix[offset_kernel + w];
        }
    }
    
    return pixel;
}

/**
 * Applies a 3x3 convolution matrix to a pixel using the GPU.
 */
__device__ float gpu_applyFilter(float *image, int stride, float *matrix, int filter_dim){  
    //Cada hilo llama a este procedimiento por cada pixel, así que esta convolución se serializa para cada pixel pero se paraleliza por el metodo anterior

    float pixel = 0.0f;
   
    for (int h = 0; h < filter_dim; h++)
    {
        int offset        = h * stride;
        int offset_kernel = h * filter_dim;
        
        for (int w = 0; w < filter_dim; w++)
        {
            pixel += image[offset + w] * matrix[offset_kernel + w];
        }
    }
    
    return pixel;
}

/**
 * Applies a Gaussian 3x3 filter to a given image using the CPU.
 */
void cpu_gaussian(int ancho, int alto, float *image, float *image_out){
    float gaussian[9] = { 1.0f / 16.0f, 2.0f / 16.0f, 1.0f / 16.0f,
                          2.0f / 16.0f, 4.0f / 16.0f, 2.0f / 16.0f,
                          1.0f / 16.0f, 2.0f / 16.0f, 1.0f / 16.0f };
    
    for (int h = 0; h < (alto - 2); h++)
    {
        int offset_t = h * ancho;
        int offset   = (h + 1) * ancho;
        
        for (int w = 0; w < (ancho - 2); w++)
        {
            image_out[offset + (w + 1)] = cpu_applyFilter(&image[offset_t + w],
                                                          ancho, gaussian, 3);
        }
    }
}

/**
 * Applies a Gaussian 3x3 filter to a given image using the GPU. Versión CON implementación de SHARED MEMORY
 */
__global__ void gpu_gaussian(int ancho, int alto, float *image, float *image_out){
    float gaussian[9] = { 1.0f / 16.0f, 2.0f / 16.0f, 1.0f / 16.0f,
                          2.0f / 16.0f, 4.0f / 16.0f, 2.0f / 16.0f,
                          1.0f / 16.0f, 2.0f / 16.0f, 1.0f / 16.0f };

    __shared__ float sh_block[BLOCK_SIZE_SH * BLOCK_SIZE_SH];                      
    
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if(x < ancho && y < alto){ // Está dentro de los limites de la imagen
        sh_block[threadIdx.y*BLOCK_SIZE_SH + threadIdx.x] = image[y*ancho+x]; //Cada bloque de shared memory se matchea con un bloque y/x y cada thread copia su pixel asignado al bloque de SM. Como la shared es compartida solo entre bloques de threads no se "pisan" entre distintos bloques.
    }

    __syncthreads(); //Para asegurarme que todos los threads del bloque copiaron su pixel al arreglo de la SharedMemory

    if (x < (ancho - 2) && y < (alto - 2)) 
    {
        int offset_t = y * ancho + x;
        int offset   = (y + 1) * ancho + (x + 1);
        
        image_out[offset] = gpu_applyFilter(&sh_block[offset_t],
                                            ancho, gaussian, 3);
    }
}

/**
 * Applies a Gaussian 3x3 filter to a given image using the GPU. Versión SIN implementación de SHARED MEMORY
 */
__global__ void gpu_gaussian1(int ancho, int alto, float *image, float *image_out){
    float gaussian[9] = { 1.0f / 16.0f, 2.0f / 16.0f, 1.0f / 16.0f,
                          2.0f / 16.0f, 4.0f / 16.0f, 2.0f / 16.0f,
                          1.0f / 16.0f, 2.0f / 16.0f, 1.0f / 16.0f };
    
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (x < (ancho - 2) && y < (alto - 2)) 
    {
        int offset_t = y * ancho + x;
        int offset   = (y + 1) * ancho + (x + 1);
        
        image_out[offset] = gpu_applyFilter(&image[offset_t],
                                            ancho, gaussian, 3);
    }
}

/**
 * Calculates the gradient of an image using a Sobel filter on the CPU.
 */
void cpu_sobel(int ancho, int alto, float *image, float *image_out){
    float sobel_x[9] = { 1.0f,  0.0f, -1.0f,
                         2.0f,  0.0f, -2.0f,
                         1.0f,  0.0f, -1.0f };
    float sobel_y[9] = { 1.0f,  2.0f,  1.0f,
                         0.0f,  0.0f,  0.0f,
                        -1.0f, -2.0f, -1.0f };
    
    for (int h = 0; h < (alto - 2); h++)
    {
        int offset_t = h * ancho;
        int offset   = (h + 1) * ancho;
        
        for (int w = 0; w < (ancho - 2); w++)
        {
            float gx = cpu_applyFilter(&image[offset_t + w], ancho, sobel_x, 3);
            float gy = cpu_applyFilter(&image[offset_t + w], ancho, sobel_y, 3);
            
            // Note: The output can be negative or exceed the max. color value
            // of 255. We compensate this afterwards while storing the file.
            image_out[offset + (w + 1)] = sqrtf(gx * gx + gy * gy);
        }
    }
}

/**
 * Calculates the gradient of an image using a Sobel filter on the GPU. Versión CON implementación de SHARED MEMORY
 */
__global__ void gpu_sobel(int ancho, int alto, float *image, float *image_out){
    float sobel_x[9] = { 1.0f,  0.0f, -1.0f,
                         2.0f,  0.0f, -2.0f,
                         1.0f,  0.0f, -1.0f };
    float sobel_y[9] = { 1.0f,  2.0f,  1.0f,
                         0.0f,  0.0f,  0.0f,
                        -1.0f, -2.0f, -1.0f };

    __shared__ float sh_block[BLOCK_SIZE_SH * BLOCK_SIZE_SH];
    
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int x = blockIdx.x * blockDim.x + threadIdx.x;

    if(x < ancho && y < alto){ // Está dentro de los limites de la imagen
        sh_block[threadIdx.y*BLOCK_SIZE_SH + threadIdx.x] = image[y*ancho+x]; //Cada bloque de shared memory se matchea con un bloque y/x y cada thread copia su pixel asignado al bloque de SM. Como la shared es compartida solo entre bloques de threads no se "pisan" entre distintos bloques.
    }

    __syncthreads(); //Para asegurarme que todos los threads del bloque copiaron su pixel al arreglo de la SharedMemory

    if(y<alto-2 && x< ancho-2){

        int offset_t = y * ancho;
        int offset   = (y + 1) * ancho;
        
        float gx = gpu_applyFilter(&sh_block[offset_t + x], ancho, sobel_x, 3);
        float gy = gpu_applyFilter(&sh_block[offset_t + x], ancho, sobel_y, 3);
            
            // Note: The output can be negative or exceed the max. color value
            // of 255. We compensate this afterwards while storing the file.
        image_out[offset + (x + 1)] = sqrtf(gx * gx + gy * gy);
        
    }
}

/**
 * Calculates the gradient of an image using a Sobel filter on the GPU. Versión SIN implementación de SHARED MEMORY
 */
__global__ void gpu_sobel1(int ancho, int alto, float *image, float *image_out){
    float sobel_x[9] = { 1.0f,  0.0f, -1.0f,
                         2.0f,  0.0f, -2.0f,
                         1.0f,  0.0f, -1.0f };
    float sobel_y[9] = { 1.0f,  2.0f,  1.0f,
                         0.0f,  0.0f,  0.0f,
                        -1.0f, -2.0f, -1.0f };
    
    int h= blockIdx.y * blockDim.y + threadIdx.y;
    int w = blockIdx.x * blockDim.x + threadIdx.x;

    if(h<alto-2 && w< ancho-2){

        int offset_t = h * ancho;
        int offset   = (h + 1) * ancho;
        
        float gx = gpu_applyFilter(&image[offset_t + w], ancho, sobel_x, 3);
        float gy = gpu_applyFilter(&image[offset_t + w], ancho, sobel_y, 3);
            
            // Note: The output can be negative or exceed the max. color value
            // of 255. We compensate this afterwards while storing the file.
        image_out[offset + (w + 1)] = sqrtf(gx * gx + gy * gy);
        
    }
}

int main(int argc, char **argv){
    BMPImage bitmap          = { 0 };
    float    *d_bitmap       = { 0 };
    float    *image_out[2]   = { 0 };
    float    *d_image_out[2] = { 0 };
    int      image_size      = 0;
    tval     t[2]            = { 0 };
    double   elapsed[2]      = { 0 };
    dim3     grid(1);                       // The grid will be defined later
    dim3     block(BLOCK_SIZE, BLOCK_SIZE); // The block size will not change
    
    // Make sure the filename is provided
    if (argc != 2){
        fprintf(stderr, "Error: The filename is missing!\n");
        return -1;
    }
    
    // Read the input image and update the grid dimension
    bitmap     = readBMP(argv[1]);
    image_size = bitmap.ancho * bitmap.alto;
    grid       = dim3(((bitmap.ancho  + (BLOCK_SIZE - 1)) / BLOCK_SIZE),
                      ((bitmap.alto + (BLOCK_SIZE - 1)) / BLOCK_SIZE));
    
    printf("Image opened (ancho=%d alto=%d).\n", bitmap.ancho, bitmap.alto);
    
    // Allocate the intermediate image buffers for each step
    for (int i = 0; i < 2; i++){
        image_out[i] = (float *)calloc(image_size, sizeof(float));
        
        hipMalloc(&d_image_out[i], image_size * sizeof(float));
        hipMemset(d_image_out[i], 0, image_size * sizeof(float));
    }

    hipMalloc(&d_bitmap, image_size * sizeof(float) * 3);
    hipMemcpy(d_bitmap, bitmap.data,
               image_size * sizeof(float) * 3, hipMemcpyHostToDevice);
    
    // Step 1: Convert to grayscale
    {
        //Launch the CPU version
        gettimeofday(&t[0], NULL);
        cpu_grayscale(bitmap.ancho, bitmap.alto, bitmap.data, image_out[0]);
        gettimeofday(&t[1], NULL);
        
        elapsed[0] = get_elapsed(t[0], t[1]);
        
        // Launch the GPU version
        gettimeofday(&t[0], NULL);
        gpu_grayscale<<<grid, block>>>(bitmap.ancho, bitmap.alto,
                                        d_bitmap, d_image_out[0]);
        
        hipMemcpy(image_out[0], d_image_out[0],
                    image_size * sizeof(float), hipMemcpyDeviceToHost);
        gettimeofday(&t[1], NULL);
        
        elapsed[1] = get_elapsed(t[0], t[1]);
        
        // Store the result image in grayscale
        store_result(1, elapsed[0], elapsed[1], bitmap.ancho, bitmap.alto, image_out[0]);
    }
    
    // Step 2: Apply a 3x3 Gaussian filter
    {
        // Launch the CPU version
        gettimeofday(&t[0], NULL);
        cpu_gaussian(bitmap.ancho, bitmap.alto, image_out[0], image_out[1]);
        gettimeofday(&t[1], NULL);
        
        elapsed[0] = get_elapsed(t[0], t[1]);
        
        // Launch the GPU version
        gettimeofday(&t[0], NULL);
        gpu_gaussian<<<grid, block>>>(bitmap.ancho, bitmap.alto,
                                      d_image_out[0], d_image_out[1]);
        
        hipMemcpy(image_out[1], d_image_out[1],
                   image_size * sizeof(float), hipMemcpyDeviceToHost);
        gettimeofday(&t[1], NULL);
        
        elapsed[1] = get_elapsed(t[0], t[1]);
        
        // Store the result image with the Gaussian filter applied
        store_result(2, elapsed[0], elapsed[1], bitmap.ancho, bitmap.alto, image_out[1]);
    }
    

    // Step 3: Apply a Sobel filter
    {
        // Launch the CPU version
        gettimeofday(&t[0], NULL);
        cpu_sobel(bitmap.ancho, bitmap.alto, image_out[1], image_out[0]);
        gettimeofday(&t[1], NULL);
        
        elapsed[0] = get_elapsed(t[0], t[1]);
        
        // Launch the GPU version
        gettimeofday(&t[0], NULL);
        gpu_sobel<<<grid, block>>>(bitmap.ancho, bitmap.alto,
                                   d_image_out[1], d_image_out[0]);
        
        hipMemcpy(image_out[0], d_image_out[0],
                   image_size * sizeof(float), hipMemcpyDeviceToHost);
        gettimeofday(&t[1], NULL);
        
        elapsed[1] = get_elapsed(t[0], t[1]);
        
        // Store the final result image with the Sobel filter applied
        store_result(3, elapsed[0], elapsed[1], bitmap.ancho, bitmap.alto, image_out[0]);
    } 
    // Release the allocated memory
    for (int i = 0; i < 2; i++)
    {
        free(image_out[i]);
        hipFree(d_image_out[i]);
    }
    
    freeBMP(bitmap);
    hipFree(d_bitmap);
    
    return 0;
}
