#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <sys/time.h>
#include <unistd.h>

#include "openmp_functions.c"
#include "cpu_functions.cu"
#include "gpu_functions.cu"

#define BLOCK_SIZE  16
#define HEADER_SIZE 138
#define bold        "\x1B[1m"     
#define normal      "\x1B[0m"

typedef unsigned char BYTE;

/**
 * Structure that represents a BMP image.
 */
typedef struct {
    int   ancho;
    int   alto;
    float *data;
} BMPImage;

typedef struct timeval tval;

BYTE g_info[HEADER_SIZE]; // Reference header

/**
 * Reads a BMP 24bpp file and returns a BMPImage structure.
 * Thanks to https://stackoverflow.com/a/9296467
 */
BMPImage readBMP(char *filename){
    BMPImage bitmap = { 0 };
    int      size   = 0;
    BYTE     *data  = NULL;
    FILE     *file  = fopen(filename, "rb");
    
    // Read the header (expected BGR - 24bpp)
    if(!fread(g_info, sizeof(BYTE), HEADER_SIZE, file)){
        exit(1);
    }

    // Get the image ancho / alto from the header
    bitmap.ancho  = *((int *)&g_info[18]);
    bitmap.alto = *((int *)&g_info[22]);
    size          = *((int *)&g_info[34]);
    
    // Read the image data
    data = (BYTE *)malloc(sizeof(BYTE) * size);
    if(!fread(data, sizeof(BYTE), size, file)){
        exit(1);
    }

    // Convert the pixel values to float
    bitmap.data = (float *)malloc(sizeof(float) * size);
    
    for (int i = 0; i < size; i++)
    {
        bitmap.data[i] = (float)data[i];
    }
    
    fclose(file);
    free(data);
    
    return bitmap;
}

/**
 * Writes a BMP file in grayscale given its image data and a filename.
 */
void writeBMPGrayscale(int ancho, int alto, float *image, char *filename){
    FILE *file = NULL;
    
    file = fopen(filename, "wb");
    
    // Write the reference header
    fwrite(g_info, sizeof(BYTE), HEADER_SIZE, file);
    
    // Unwrap the 8-bit grayscale into a 24bpp (for simplicity)
    for (int h = 0; h < alto; h++) {
        int offset = h * ancho;

        for (int w = 0; w < ancho; w++){
            BYTE pixel = (BYTE)((image[offset + w] > 255.0f) ? 255.0f :
                                (image[offset + w] < 0.0f)   ? 0.0f   :
                                                               image[offset + w]);
            
            // Repeat the same pixel value for BGR
            fputc(pixel, file);
            fputc(pixel, file);
            fputc(pixel, file);
        }
    }
    
    fclose(file);
}

/**
 * Releases a given BMPImage.
 */
void freeBMP(BMPImage bitmap){
    free(bitmap.data);
}

/**
 * Checks if there has been any CUDA error. The method will automatically print
 * some information and exit the program when an error is found.
 */
void checkCUDAError(){
    hipError_t hipError_t = hipGetLastError();
    
    if(hipError_t != hipSuccess) {
        printf("CUDA Error: Returned %d: %s\n", hipError_t,
                                                hipGetErrorString(hipError_t));
        exit(-1);
    }
}

/**
 * Calculates the elapsed time between two time intervals (in milliseconds).
 */
double get_elapsed(tval t0, tval t1){
    return (double)(t1.tv_sec - t0.tv_sec) * 1000.0L + (double)(t1.tv_usec - t0.tv_usec) / 1000.0L;
}

/**
 * Stores the result image and prints a message.
 */
void guardar_resultado(int index, double tiempo_cpu, double tiempo_gpu_G, double tiempo_gpu_S,
                    double tiempo_openmp, int ancho, int alto, float *image_cuda, float *image_omp){
    char path1[255];
    char path2[255];
    //char path3[255];
    

    sprintf(path1, "images/%d_omp.bmp", index);
    writeBMPGrayscale(ancho, alto, image_omp, path1);
    sprintf(path2, "images/%d_cuda.bmp", index);
    writeBMPGrayscale(ancho, alto, image_cuda, path2);
    
    printf("\nPaso #%d Completado:",index);
    switch(index){
        case 1:
            printf(" Escala de Grises\n");
            break;
        case 2:
            printf(" Blur de Gauss\n");
            break;
        case 3:
            printf(" Filtro de Sobel\n");
            break;
    }
    printf("    Tiempo en CPU:          %fms\n",tiempo_cpu);
    
    printf("    Tiempo en OpenMP:       %fms\n",tiempo_openmp);
    printf("        Speedup: %.2f%%\n",(tiempo_cpu/tiempo_openmp -1)*100);

    if (tiempo_gpu_G == 0) {
        printf("    [Versión en GPU no disponible]\n");
    }
    else if(tiempo_gpu_G==tiempo_gpu_S){
        printf("    Tiempo en GPU:          %fms\n",tiempo_gpu_G);
        printf("        Speedup: %.2f%%\n",(tiempo_cpu/tiempo_gpu_G -1)*100);
        return;
    }
    else{
        printf("    Tiempo en GPU (Global): %fms\n",tiempo_gpu_G);
        printf("        Speedup: %.2f%%\n",(tiempo_cpu/tiempo_gpu_G -1)*100);
    }
    if(tiempo_gpu_S!=0){
        printf("    Tiempo en GPU (Shared): %fms\n",tiempo_gpu_S);
        printf("        Speedup: %.2f%%\n",(tiempo_cpu/tiempo_gpu_S -1)*100);
    }
    else{
        printf("    [Versión en GPU (Shared Memory) no disponible]\n");
    }
}

int main(int argc, char **argv){
    BMPImage bitmap          = { 0 };
    float    *d_bitmap       = { 0 };
    float    *image_cuda1[2]   = { 0 };
    //float    *image_cuda2[2]   = { 0 };
    float    *image_omp[2]   = { 0 };
    float    *d1_image_out[2] = { 0 };
    float    *d2_image_out[2] = { 0 };
    int      image_size      =   0;
    tval     t[2]            = { 0 };
    double   elapsed[4]      = { 0 };
    double   suma[4]         = { 0 };
    int      threads         =  16;
    dim3     grid(1);                       // The grid will be defined later
    dim3     block(BLOCK_SIZE, BLOCK_SIZE); // The block size will not change
    
    // Make sure the filename is provided
    if (argc == 1){
        fprintf(stderr, "Error: The filename is missing!\n");
        return -1;
    }
    else if(argc==3){ //Si me pasan la cantidad de threads a utilizar
        threads=atoi(argv[2]);
    }
    if(access(argv[1],F_OK)==-1){ // Chequeo si la imagen a abrir existe
        printf("%sPath '%s' inválido%s - Intente nuevamente\n",bold,argv[1],normal);
        return -1;
    }
    
    // Read the input image and update the grid dimension
    bitmap     = readBMP(argv[1]);
    image_size = bitmap.ancho * bitmap.alto;
    grid       = dim3(((bitmap.ancho  + (BLOCK_SIZE - 1)) / BLOCK_SIZE),
                      ((bitmap.alto   + (BLOCK_SIZE - 1)) / BLOCK_SIZE));

    printf("Imagen '%s' abierta (Ancho = %dp - Alto = %dp) | ",argv[1]+9,bitmap.ancho, bitmap.alto);
    printf("Número de hilos: %d\n",threads);

    // Allocate the intermediate image buffers for each step
    for (int i = 0; i < 2; i++){
        image_cuda1[i] = (float *)calloc(image_size, sizeof(float));
        //image_cuda2[i] = (float *)calloc(image_size, sizeof(float));
        image_omp[i] = (float *)calloc(image_size, sizeof(float));
        hipMalloc(&d1_image_out[i], image_size * sizeof(float));
        hipMemset(d1_image_out[i], 0, image_size * sizeof(float));
        hipMalloc(&d2_image_out[i], image_size * sizeof(float));
        hipMemset(d2_image_out[i], 0, image_size * sizeof(float));
    }

    hipMalloc(&d_bitmap, image_size * sizeof(float) * 3);
    hipMemcpy(d_bitmap, bitmap.data,
               image_size * sizeof(float) * 3, hipMemcpyHostToDevice);
    
    // Step 1: Convert to grayscale
    {
        //Launch the CPU version
        gettimeofday(&t[0], NULL);
        cpu_grayscale(bitmap.ancho, bitmap.alto, bitmap.data, image_omp[0]);
        gettimeofday(&t[1], NULL);        
        elapsed[0] = get_elapsed(t[0], t[1]);

        //Launch the OpenMP version
        gettimeofday(&t[0], NULL);
        openmp_grayscale(bitmap.ancho, bitmap.alto, bitmap.data, image_omp[0],threads);
        gettimeofday(&t[1], NULL);
        elapsed[3] = get_elapsed(t[0], t[1]);

        // Launch the GPU version
        gettimeofday(&t[0], NULL);
        gpu_grayscale<<<grid, block>>>(bitmap.ancho, bitmap.alto,d_bitmap, d1_image_out[0]);
        hipMemcpy(image_cuda1[0], d1_image_out[0],image_size * sizeof(float), hipMemcpyDeviceToHost);
        gettimeofday(&t[1], NULL);
        elapsed[1] = get_elapsed(t[0], t[1]);
        elapsed[2] = elapsed[1];
        
        //Hago el mismo procedimiento para la imagen que usa SM
        gpu_grayscale<<<grid, block>>>(bitmap.ancho, bitmap.alto,d_bitmap, d2_image_out[0]); 
        //hipMemcpy(image_cuda2[0], d1_image_out[0],image_size * sizeof(float), hipMemcpyDeviceToHost);
        
        // Store the result image in grayscale
        guardar_resultado(1, elapsed[0], elapsed[1], elapsed[2], elapsed[3], bitmap.ancho, bitmap.alto, image_cuda1[0],image_omp[0]);
        suma[0]+=elapsed[0];suma[1]+=elapsed[1];suma[2]+=elapsed[2];suma[3]+=elapsed[3];   
    }

    // Step 2: Apply a 3x3 Gaussian filter
    {
        // Launch the CPU version
        gettimeofday(&t[0], NULL);
        cpu_gaussian(bitmap.ancho, bitmap.alto, image_omp[0], image_omp[1]);
        gettimeofday(&t[1], NULL);
        elapsed[0] = get_elapsed(t[0], t[1]);
        
        //Launch the OpenMP version
        gettimeofday(&t[0], NULL);
        openmp_gaussian(bitmap.ancho, bitmap.alto, image_omp[0], image_omp[1],threads);
        gettimeofday(&t[1], NULL);
        elapsed[3] = get_elapsed(t[0], t[1]);

        // Launch the GPU-GM version
        gettimeofday(&t[0], NULL);
        gpu_gaussian_GM<<<grid, block>>>(bitmap.ancho, bitmap.alto,d1_image_out[0], d1_image_out[1]);
        hipMemcpy(image_cuda1[1], d1_image_out[1],image_size * sizeof(float), hipMemcpyDeviceToHost);
        gettimeofday(&t[1], NULL);
        elapsed[1] = get_elapsed(t[0], t[1]);

        // Launch the GPU-SM version
        gettimeofday(&t[0], NULL);
        gpu_gaussian_SM<<<grid, block>>>(bitmap.ancho, bitmap.alto,d2_image_out[0], d2_image_out[1]);
        hipMemcpy(image_cuda1[1], d2_image_out[1],image_size * sizeof(float), hipMemcpyDeviceToHost);
        gettimeofday(&t[1], NULL);
        elapsed[2] = get_elapsed(t[0], t[1]);
        
        
        // Store the result image with the Gaussian filter applied
        guardar_resultado(2, elapsed[0], elapsed[1], elapsed[2], elapsed[3], bitmap.ancho, bitmap.alto, image_cuda1[1],image_omp[1]);
        suma[0]+=elapsed[0];suma[1]+=elapsed[1];suma[2]+=elapsed[2];suma[3]+=elapsed[3];   
    }
    

    // Step 3: Apply a Sobel filter
    {
        // Launch the CPU version
        gettimeofday(&t[0], NULL);
        cpu_sobel(bitmap.ancho, bitmap.alto, image_omp[1], image_omp[0]);
        gettimeofday(&t[1], NULL);
        elapsed[0] = get_elapsed(t[0], t[1]);
        
        //Launch the OpenMP version
        gettimeofday(&t[0], NULL);
        openmp_sobel(bitmap.ancho, bitmap.alto, image_omp[1],image_omp[0],threads);
        gettimeofday(&t[1], NULL);
        elapsed[3] = get_elapsed(t[0], t[1]);

        // Launch the GPU-GM version
        gettimeofday(&t[0], NULL);
        gpu_sobel_GM<<<grid, block>>>(bitmap.ancho, bitmap.alto,d1_image_out[1], d1_image_out[0]);
        hipMemcpy(image_cuda1[0], d1_image_out[0],image_size * sizeof(float), hipMemcpyDeviceToHost);
        gettimeofday(&t[1], NULL);
        elapsed[1] = get_elapsed(t[0], t[1]);
        
        // Launch the GPU-SM version
        gettimeofday(&t[0], NULL);
        gpu_sobel_SM<<<grid, block>>>(bitmap.ancho, bitmap.alto,d2_image_out[1], d2_image_out[0]);
        hipMemcpy(image_cuda1[0], d2_image_out[0],image_size * sizeof(float), hipMemcpyDeviceToHost);
        gettimeofday(&t[1], NULL);
        elapsed[2] = get_elapsed(t[0], t[1]);

        // Store the final result image with the Sobel filter applied
        guardar_resultado(3, elapsed[0], elapsed[1], elapsed[2], elapsed[3], bitmap.ancho, bitmap.alto, image_cuda1[0],image_omp[0]);
        suma[0]+=elapsed[0];suma[1]+=elapsed[1];suma[2]+=elapsed[2];suma[3]+=elapsed[3];
    }
    
    printf("\nTiempo total en ejecución secuencial:             %.3fms\n",suma[0]);
    printf("\nTiempo total usando paralelismo de OpenMP:        %.3fms\n",suma[3]);
    printf("    (Speedup total de %.2f%%)\n",(suma[0]/suma[3] -1)*100);
    printf("\nTiempo total usando paralelismo de CUDA (Global): %.3fms\n",suma[1]);
    printf("    (Speedup total de %2.5f%%)\n",(suma[0]/suma[1] -1)*100);
    //printf("\n[Versión en GPU (Shared Memory) no disponible]\n");
    printf("\nTiempo total usando paralelismo de CUDA (Shared): %.3fms\n",suma[2]);
    printf("    (Speedup total de %2.5f%%)\n",(suma[0]/suma[2] -1)*100);

    // Release the allocated memory
    for (int i = 0; i < 2; i++){
        free(image_cuda1[i]);
        //free(image_cuda2[i]);
        free(image_omp[i]);
        hipFree(d1_image_out[i]);
        hipFree(d2_image_out[i]);
    }
    
    freeBMP(bitmap);
    hipFree(d_bitmap);
    printf("\n");
    return 0;
}

